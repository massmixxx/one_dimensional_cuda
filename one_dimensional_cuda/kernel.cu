#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include ""
#include <bitset>
#include "kernel.h"
#include "dev_array.h"
#include <stdio.h>
#include <vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <chrono>
#include "BitMapSaver.h"


using namespace std;
using namespace thrust;



__global__ void countAutomatKernel(unsigned char *result, unsigned int size, unsigned char RULE, int CYCLE)
{
	size_t t = threadIdx.x + (blockDim.x * blockIdx.x);
	if (t > size) return;

	unsigned int location = size * CYCLE + t;
	unsigned int previousRow = location-size;
	
	if (t == 0) {
		result[location] = (RULE & 1 << (7 & (result[previousRow + 1] << 2 | result[previousRow] << 1 | result[location - 1]))) ? 1 : 0;
		return;
	}
	if (t == size - 1) {
		result[location] = (RULE & 1 << (7 & (result[previousRow - size +1] << 2 | result[previousRow] << 1 | result[previousRow- 1]))) ? 1 : 0;
		return;
	}
	result[location] = (RULE & 1 << (7 & (result[previousRow - 1] << 2 | result[previousRow] << 1 | result[previousRow + 1]))) ? 1 : 0;
}

kernel::kernel() { }
kernel::~kernel() {}

int kernel::printCUDA(bool isImage, unsigned char RULE,unsigned int CYCLES,unsigned int DIMENSION)
{
	cout << "IN printCUDA" << endl;

	int threadsPerBlock = DIMENSION;
	unsigned long long blocksPerGrid = 1;
	if (DIMENSION > 1024) {
		threadsPerBlock = 1024;
		blocksPerGrid = (DIMENSION / threadsPerBlock) + 1;
	}

	unsigned char *result = new unsigned char[DIMENSION*CYCLES];
	
	for (unsigned int i = 0; i < DIMENSION; i++)
		result[i] = 0;
		
	result[(DIMENSION / 2)] = 1;
	dev_array<unsigned char> device_A(DIMENSION*CYCLES);
	device_A.set(result, DIMENSION*CYCLES);

	auto chrono_time_start = chrono::system_clock::now();

	for (unsigned int i = 1; i < CYCLES; i++) {
		countAutomatKernel << <blocksPerGrid, threadsPerBlock >> > (device_A.getData(), DIMENSION, RULE, i);		
	}

	auto chrono_time_end = chrono::system_clock::now();

	hipDeviceSynchronize();
	device_A.get(result, DIMENSION*CYCLES); 

	if (isImage)
		BitMapSaver::SaveBitmap(result, DIMENSION, CYCLES, "GPUImageResult");

	device_A.~dev_array();
	delete result;

	auto elapsed = chrono::duration_cast<chrono::milliseconds>(chrono_time_end - chrono_time_start);
	auto elapsedMicro = chrono::duration_cast<chrono::microseconds>(chrono_time_end - chrono_time_start);
	cout << ">GPU< Czas chrono: " << elapsed.count() << "milisekund. |  "<< elapsedMicro.count() << " microsekund." << endl << endl;

	return 0;
}



